#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */
#include "search.h"

/*
 * CUDA Kernel Device code
 *
 * Search passed data set for a float value and if the value is at the thread index set the foundIndex value
 */
__global__ void search(int *d_d, int *d_i, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        int inputVal = d_d[i];
        if(inputVal == d_v)
        {
            d_i[0] = i;
        }
    }
}

__host__ int * allocateRandomHostMemory(int numElements)
{
    srand(time(0));
    size_t size = numElements * sizeof(int);

    // Allocate the host input vector A
    int *h_d = (int *)malloc(size);

    // Verify that allocations succeeded
    if (h_d == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_d[i] = rand();
    }

    return h_d;
}

// Based heavily on https://www.gormanalysis.com/blog/reading-and-writing-csv-files-with-cpp/
// Presumes that there is no header in the csv file
__host__ std::tuple<int * , int>readCsv(std::string filename)
{
    std::vector<int> tempResult;
    // Create an input filestream
    std::ifstream myFile(filename);

    // Make sure the file is open
    if(!myFile.is_open()) throw std::runtime_error("Could not open file");

    // Helper vars
    std::string line, colname;
    int val;

    // Read data, line by line
    while(std::getline(myFile, line))
    {
        // Create a stringstream of the current line
        std::stringstream ss(line);
        
        // Extract each integer
        while(ss >> val){
            tempResult.push_back(val);
            // If the next token is a comma, ignore it and move on
            if(ss.peek() == ',') ss.ignore();
        }
    }

    // Close file
    myFile.close();
    int numElements = tempResult.size();
    int result[numElements];
    // Copy all elements of vector to array
    std::copy(tempResult.begin(), tempResult.end(), result);

    return {result, numElements};
}

__host__ std::tuple<int *, int *> allocateDeviceMemory(int numElements)
{
    // Allocate the device input vector A
    int *d_d = NULL;
    size_t size = numElements * sizeof(int);
    hipError_t err = hipMalloc(&d_d, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int *d_i;
    hipMalloc((void**)&d_i, sizeof(int));

    return {d_d, d_i};
}

__host__ void copyFromHostToDevice(int h_v, int *h_d, int h_i, int *d_d, int *d_i, int numElements)
{
    size_t size = numElements * sizeof(int);

    hipError_t err = hipMemcpy(d_d, h_d, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(d_v), &h_v, sizeof(int), 0, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy constant int d_v from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_i, &h_i, sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy int d_i from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void executeKernel(int *d_d, int *d_i, int numElements, int threadsPerBlock)
{
    // Launch the search CUDA Kernel
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    search<<<blocksPerGrid, threadsPerBlock>>>(d_d, d_i, numElements);
    hipError_t err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void copyFromDeviceToHost(int *d_i, int &h_i)
{
    // Copy the device result int (found index) in device memory to the host result int
    // in host memory.
    hipError_t err = hipMemcpy(&h_i, d_i, sizeof(int), hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy int d_i from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


// Free device global memory
__host__ void deallocateMemory(int *h_d, int *d_d, int *d_i)
{

    hipError_t err = hipFree(d_d);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector d_d (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_i);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device int variable d_i (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

// Reset the device and exit
__host__ void cleanUpDevice()
{
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

__host__ void outputToFile(std::string currentPartId, int *data, int numElements, int searchValue, int foundIndex)
{
	std::string outputFileName = "output-" + currentPartId + ".txt";
	// NOTE: Do not remove this output to file statement as it is used to grade assignment,
	// so it should be called by each thread
	std::ofstream outputFile;
	outputFile.open (outputFileName, std::ofstream::app);

    outputFile << "Data: ";
    for (int i = 0; i < numElements; ++i)
        outputFile << data[i] << " ";
    outputFile << "\n";
    outputFile << "Searching for value: " << searchValue << "\n";
	outputFile << "Found Index: " << foundIndex << "\n";

	outputFile.close();
}

__host__ std::tuple<int, int, std::string, int, std::string, bool> parseCommandLineArguments(int argc, char *argv[])
{
    int numElements = 10;
    int h_v = -1;
    int threadsPerBlock = 256;
    std::string currentPartId = "test";
    bool sortInputData = true;
    std::string inputFilename = "NULL";

    for(int i = 1; i < argc; i++)
    {
        std::string option(argv[i]);
        i++;
        std::string value(argv[i]);
        if(option.compare("-s") == 0)
        {
            if(value == "false")
            {
                sortInputData = false;
            }
        }
        else if(option.compare("-t") == 0) 
        {
            threadsPerBlock = atoi(value.c_str());
        }
        else if(option.compare("-n") == 0) 
        {
            numElements = atoi(value.c_str());
        }
        else if(option.compare("-v") == 0) 
        {
            h_v = atoi(value.c_str());
        }
        else if(option.compare("-f") == 0) 
        {
            inputFilename = value;
        }
        else if(option.compare("-p") == 0) 
        {
            currentPartId = value;
        }
    }

    return {numElements, h_v, currentPartId, threadsPerBlock, inputFilename, sortInputData};
}

__host__ std::tuple<int *, int, int> setUpSearchInput(std::string inputFilename, int numElements, int h_v, bool sortInputData)
{
    srand(time(0));
    int *h_d;

    if(inputFilename.compare("NULL") != 0)
    {
        tuple<int *, int>csvData = readCsv(inputFilename);
        h_d = get<0>(csvData);
        numElements = get<1>(csvData);
    }
    else 
    {
        h_d = allocateRandomHostMemory(numElements);
    }

    if(sortInputData)
    {
        sort(h_d, h_d + numElements);
    }

    if(h_v == -1)
    {
        // Roll a 6-sided die if not a 6 generate from a random value in the input data otherwise pick a random value
        int diceRoll = rand()%6;
        h_v = diceRoll < 5 ? h_d[rand()%numElements] : rand();
    }

    return {h_d, numElements, h_v};
}

/*
 * Host main routine
 * -s true|false - sort data prior to search
 * -n numElements - the number of elements of random data to create
 * -v searchValue - the value to search for in the data
 * -f inputFile - the file for non-random input data
 * -p currentPartId - the Coursera Part ID
 * -t threadsPerBlock - the number of threads to schedule for concurrent processing
 */
int main(int argc, char *argv[])
{
    int h_i = -1;
    int * h_d;
    
    auto[numElements, h_v, currentPartId, threadsPerBlock, inputFilename, sortInputData] = parseCommandLineArguments(argc, argv);
    std::tuple<int *, int, int> searchInputTuple = setUpSearchInput(inputFilename, numElements, h_v, sortInputData);

    h_d = get<0>(searchInputTuple);
    numElements = get<1>(searchInputTuple);
    h_v = get<2>(searchInputTuple);

    auto[d_d, d_i] = allocateDeviceMemory(numElements);
    copyFromHostToDevice(h_v, h_d, h_i, d_d, d_i, numElements);

    executeKernel(d_d, d_i, numElements, threadsPerBlock);

    copyFromDeviceToHost(d_i, h_i);
    outputToFile(currentPartId, h_d, numElements, h_v, h_i);

    
    deallocateMemory(h_d, d_d, d_i);

    cleanUpDevice();
    return 0;
}